#include "hip_stream.h"
#include "error.h"

hipStream_t create_stream()
{
    hipStream_t stream = {};
    assert_cuda_ok(hipStreamCreate(&stream));
    return stream;
}

CudaStream::CudaStream() : CudaStream(create_stream())
{
}

void CudaStream::synchronize()
{
    assert_cuda_ok(hipStreamSynchronize(static_cast<hipStream_t>(native())));
}

CudaStream::~CudaStream() noexcept
{
    if (&current() == this)
        default_stream().set_as_current();
    if (this != &default_stream())
        hipStreamDestroy(static_cast<hipStream_t>(native()));
}

CudaStream CudaStream::default_ (0);
