#include "hip_stream.h"
#include "error.h"

hipStream_t create_stream()
{
    hipStream_t stream = {};
    assert_cuda_ok(hipStreamCreate(&stream));
    return stream;
}

CudaStream::CudaStream() : CudaStream(create_stream())
{
}

void CudaStream::synchronize()
{
    assert_cuda_ok(hipStreamSynchronize(static_cast<hipStream_t>(native())));
}

CudaStream::~CudaStream() noexcept
{
    hipStreamDestroy(static_cast<hipStream_t>(native()));
}

CudaStream CudaStream::default_ (0);
